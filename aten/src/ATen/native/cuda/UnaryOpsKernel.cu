#include "hip/hip_runtime.h"
#include <limits>
#include <hip/hip_fp16.h>
#include <ATen/native/UnaryOps.h>
#include <ATen/native/cuda/Loops.cuh>
#include <ATen/Context.h>
#include <ATen/Dispatch.h>
#include <ATen/native/cuda/Loops.cuh>
#include <ATen/native/DispatchStub.h>
#include <ATen/native/TensorIterator.h>

namespace at { namespace native {

void abs_kernel_cuda(TensorIterator& iter) {
  switch (iter.dtype()) {
    case ScalarType::Bool:
      gpu_kernel(iter, []GPU_LAMBDA(bool a) { return a; });
      break;
#if __CUDA_ARCH__ >= 530
    case ScalarType::Half:
      gpu_kernel(iter, []GPU_LAMBDA(half a) -> half {
        half nega = __hneg(a);
        return __hge(a, nega) ? a : nega;
      });
      break;
#endif
    default:
      AT_DISPATCH_ALL_TYPES(iter.dtype(), "abs_cuda", [&]() {
        gpu_kernel(iter, []GPU_LAMBDA(scalar_t a) -> scalar_t {
          return std::abs(a);
        });
      });
  }
}

void bitwise_not_kernel_cuda(TensorIterator& iter) {
  if (iter.dtype() == ScalarType::Bool) {
    gpu_kernel(iter, []GPU_LAMBDA(bool a) {
      return !a;
    });
  } else {
    AT_DISPATCH_INTEGRAL_TYPES(iter.dtype(), "bitwise_not_cuda", [&]() {
      gpu_kernel(iter, []GPU_LAMBDA(scalar_t a) -> scalar_t {
        return ~a;
      });
    });
  }
}

REGISTER_DISPATCH(abs_stub, &abs_kernel_cuda);
REGISTER_DISPATCH(bitwise_not_stub, &bitwise_not_kernel_cuda);

}}
